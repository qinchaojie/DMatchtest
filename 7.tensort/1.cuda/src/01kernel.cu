#include <iostream>
#include <hip/hip_runtime.h>


__global__ void compute(float* a, float* b, float* c)
{
    int position = threadIdx.x;
    c[position] = a[position] * b[position];
}

int main()
{
    // 定义3个数组
    const int num = 3;
    float a[num] = {1, 2, 3};
    float b[num] = {5 ,7 ,9};
    float c[num] = {0};

    // 定义三个设备指针，device指针
    size_t size_array = sizeof(c);
    float* device_a = nullptr;
    float* device_b = nullptr;
    float* device_c = nullptr;

    // 分配设备空间，大小是size_array, 单位是byte
    hipMalloc(&device_a, size_array);
    hipMalloc(&device_b, size_array);
    hipMalloc(&device_c, size_array);

    // 把数据冲host复制到device,其实就是主机复制到显卡
    // 复制的是a, b
    hipMemcpy(device_a, a, size_array, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, size_array, hipMemcpyHostToDevice);

    // 执行核函数，把结果放在c上
    compute<<<1, 3>>>(device_a, device_b, device_c);

    // 把计算后的结果c复制回主机上
    hipMemcpy(c, device_c, size_array, hipMemcpyDeviceToHost);

    // 查看主机上的c内容是多少
    for (int i = 0; i < num; ++i)
        printf("c[%d] = %f\n", i, c[i]);
    return 0;
}