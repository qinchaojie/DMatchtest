
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void printHello_from_gpu(){

    printf("hello\n");
}


int main(){

    printHello_from_gpu<<<4,4>>>();
    hipDeviceSynchronize(); // 同步
    
    return 0;
}
